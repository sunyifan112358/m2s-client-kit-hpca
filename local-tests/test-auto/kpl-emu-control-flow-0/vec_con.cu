#include <iostream>
#include <hip/hip_runtime.h>
#define vect_len 120
using namespace std;

const int blocksize = 50;

// __global__ decorator signifies a kernel that can be called from the host
	
__global__ void vec_con_2(int *a, int *b, int n)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id > 5)
	{	
		int i = 0;
		do
		{
			if (id > 10)
				a[id] += b[id];
			else
			{
				if (id < 7)
					break;
				else
				{
					a[id] += 1;
					continue;
				}
			}
		} while((id + i++) < 20);
	}
}


int main(){

	const int vect_size = vect_len*sizeof(int);
	int * vect1=(int*)malloc(vect_size);
	int * vect2=(int*)malloc(vect_size);
	int * result_v1=(int*)malloc(vect_size);
	int * result_v2=(int*)malloc(vect_size);
 	bool flag;

	for(int i = 0; i < vect_len; i++)
	{
		vect1[i] = i;
		vect2[i] = 2 * i;
	}
	int *ad, *bd;
	// initialize device memory
	hipMalloc( (void**)&ad, vect_size );
	hipMalloc( (void**)&bd, vect_size );
	// copy data to device
	hipMemcpy( ad, vect1, vect_size, hipMemcpyHostToDevice );
	hipMemcpy( bd, vect2, vect_size, hipMemcpyHostToDevice );
	// setup block and grid size	
	dim3 dimBlock( blocksize, 1, 1);
	dim3 dimGrid((vect_len + blocksize - 1)/blocksize, 1 , 1);

	// call device kernel
	vec_con_2<<<dimGrid, dimBlock>>>(ad, bd, 10);
	hipMemcpy( result_v1, ad, vect_size, hipMemcpyDeviceToHost );
	hipMemcpy( result_v2, bd, vect_size, hipMemcpyDeviceToHost );

	int* a = (int*)calloc(vect_len, sizeof(int));
	int* b = (int*)calloc(vect_len, sizeof(int));

	for (int i = 0; i < vect_len; i++)
	{
		a[i] = i;
		b[i] = 2 * i;
	}

	for (int id = 0; id < vect_len; id++)
	{
		if (id > 5)
		{	
			int i = 0;
			do
			{
				if (id > 10)
					a[id] += b[id];
				else
				{
					if (id < 7)
						break;
					else
					{
						a[id] += 1;
						continue;
					}
				}
			} while((id + i++) < 20);
		}
	}

	flag = true;

	for(int i = 0; i < vect_len; i++)
	{
		if (result_v1[i] != a[i])
		{
			cout << "Test2 failed at a " << i
				<< " expecting " << a[i] 
				<< " getting " << result_v1[i]<< endl;
			flag = false;
		}
		if (result_v2[i] != b[i])
		{
			cout << "Test2 failed at b " << i
				<< " expecting " << b[i]
				<< " getting " << result_v2[i]<< endl;
			flag = false;
		}
	}

	if(flag)
		cout << "Verification test passes." <<endl;

	// free device memory
	hipFree( ad );
	hipFree( bd );
	free(vect1);
	free(vect2);
	free(result_v1);
	free(result_v2);
	return EXIT_SUCCESS;
}



