#include <iostream>
#include <hip/hip_runtime.h>
#define vect_len 120
using namespace std;

const int blocksize = 50;

// __global__ decorator signifies a kernel that can be called from the host
__global__ void vec_con_0(int *a, int *b, int n)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x ;
	for (int i = 0; i < n; i++)
	{

		if (i < 5)
		{
			if (i > 2 )
				continue;
				//break;
			else
			{
				if (id < 16)
				{
					a[id] += 1;
				}
				else
					continue;
			}
		}
		else  
		{
			if (i < 8)
			{
				if (id >15)
				{
					a[id] += 2;
				}
				else
					break;
			}
			else
				break;
		}
		a[id] += 1;
	}
}

int main(){

	const int vect_size = vect_len*sizeof(int);
	int * vect1=(int*)malloc(vect_size);
	int * vect2=(int*)malloc(vect_size);
	int * result_v1=(int*)malloc(vect_size);
	int * result_v2=(int*)malloc(vect_size);
 	bool flag;

	for(int i = 0; i < vect_len; i++)
	{
		vect1[i] = i;
		vect2[i] = 2 * i;
	}
	int *ad, *bd;
	// initialize device memory
	hipMalloc( (void**)&ad, vect_size );
	hipMalloc( (void**)&bd, vect_size );
	// copy data to device
	hipMemcpy( ad, vect1, vect_size, hipMemcpyHostToDevice );
	hipMemcpy( bd, vect2, vect_size, hipMemcpyHostToDevice );
	// setup block and grid size	
	dim3 dimBlock( blocksize, 1, 1);
	dim3 dimGrid((vect_len + blocksize - 1)/blocksize, 1 , 1);
	// call device kernel
	//vect_add<<<dimGrid, dimBlock>>>(ad, bd);
	vec_con_0<<<dimGrid, dimBlock>>>(ad, bd, 10);
	hipMemcpy( result_v1, ad, vect_size, hipMemcpyDeviceToHost );
	hipMemcpy( result_v2, bd, vect_size, hipMemcpyDeviceToHost );

	//Verify
	flag = true;

	for(int i = 0; i < vect_len; i++)
	{
		if (i < 16)
		{
			if (result_v1[i] != i + 6)
			{
				cout << " Test 0 Error at " << i << " expecting "
				<< i + 6 << " getting " << result_v1[i] <<endl;
				flag = false;
			}
			
		}
		else
		{
			if (result_v1[i] != i + 9)
			{
				cout << "Test 0 Error at " << i << " expecting "
				<< i + 9 << " getting " << result_v1[i] <<endl;
				flag = false;
			}
		}

		
	}

	if(flag)
		cout << "Verification test passes." <<endl;

	// free device memory
	hipFree( ad );
	hipFree( bd );
	free(vect1);
	free(vect2);
	free(result_v1);
	free(result_v2);
	return EXIT_SUCCESS;
}



