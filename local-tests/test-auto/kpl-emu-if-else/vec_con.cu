#include <iostream>
#include <hip/hip_runtime.h>
#define vect_len 100
using namespace std;

const int blocksize = 50;

// __global__ decorator signifies a kernel that can be called from the host
__global__ void vec_con_0(int *a, int *b, int n)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x ;

	if (id >vect_len / 2)
		a[id] += b[id];
	else
		a[id] += a[id];	
}

int main(){

	const int vect_size = vect_len*sizeof(int);
	int * vect1=(int*)malloc(vect_size);
	int * vect2=(int*)malloc(vect_size);
	int * result_v1=(int*)malloc(vect_size);
	int * result_v2=(int*)malloc(vect_size);
 	bool flag;

	for(int i = 0; i < vect_len; i++)
	{
		vect1[i] = i;
		vect2[i] = 2 * i;
	}
	int *ad, *bd;
	// initialize device memory
	hipMalloc( (void**)&ad, vect_size );
	hipMalloc( (void**)&bd, vect_size );
	// copy data to device
	hipMemcpy( ad, vect1, vect_size, hipMemcpyHostToDevice );
	hipMemcpy( bd, vect2, vect_size, hipMemcpyHostToDevice );
	// setup block and grid size	
	dim3 dimBlock( blocksize, 1, 1);
	dim3 dimGrid((vect_len + blocksize - 1)/blocksize, 1 , 1);
	// call device kernel
	//vect_add<<<dimGrid, dimBlock>>>(ad, bd);
	vec_con_0<<<dimGrid, dimBlock>>>(ad, bd, 10);
	hipMemcpy( result_v1, ad, vect_size, hipMemcpyDeviceToHost );
	hipMemcpy( result_v2, bd, vect_size, hipMemcpyDeviceToHost );

	//Verify
	flag = true;

	for(int id = 0; id < vect_len; id++)
	{
		int a = id;
		int b = 2 * id;

		if (id > vect_len / 2)
			a += b;
		else
			a += a;	

		if (a != result_v1[id])
		{
			flag = false;
			cout<< "Failed at a " << id << " expecting " 
				<< a << " getting " << result_v1[id] << endl;
		}
		if (b != result_v2[id])
		{
			flag = false;
			cout<< "Failed at b " << id << " expecting " 
				<< a << " getting " << result_v2[id] << endl;
		}
	}

	if(flag)
		cout << "Verification test passes." <<endl;

	// free device memory
	hipFree( ad );
	hipFree( bd );
	free(vect1);
	free(vect2);
	free(result_v1);
	free(result_v2);
	return EXIT_SUCCESS;
}



