#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#define vect_len 33
using namespace std;

const int blocksize = 32;

// __global__ decorator signifies a kernel that can be called from the host
__global__ void vec_con_1(int *a, int *b, int n)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < vect_len)
		for (int j = 0; j < n; j++)
		{
			if (id < vect_len / 3)
				continue;
			else
			{
				if (id < vect_len / 3 * 2)
					a[id] +=10;
				else
				{
					b[id] += 4;
					if (b[id] == 200)
						break;
				}
				a[id] += 1;
			}
			b[id] += 1;
		}
}
	
int main(){

	const int vect_size = vect_len*sizeof(int);
	int * vect1=(int*)malloc(vect_size);
	int * vect2=(int*)malloc(vect_size);
	int * result_v1=(int*)malloc(vect_size);
	int * result_v2=(int*)malloc(vect_size);
 	bool flag;

	for(int i = 0; i < vect_len; i++)
	{
		vect1[i] = i;
		vect2[i] = 2 * i;
	}
	int *ad, *bd;
	// initialize device memory
	hipMalloc( (void**)&ad, vect_size );
	hipMalloc( (void**)&bd, vect_size );
	// copy data to device
	hipMemcpy( ad, vect1, vect_size, hipMemcpyHostToDevice );
	hipMemcpy( bd, vect2, vect_size, hipMemcpyHostToDevice );
	// setup block and grid size	
	dim3 dimBlock( blocksize, 1, 1);
	dim3 dimGrid((vect_len + blocksize - 1)/blocksize, 1 , 1);

	// call device kernel
	vec_con_1<<<dimGrid, dimBlock>>>(ad, bd, 10);
	hipMemcpy( result_v1, ad, vect_size, hipMemcpyDeviceToHost );
	hipMemcpy( result_v2, bd, vect_size, hipMemcpyDeviceToHost );

	flag = true;

	for (int id = 0; id < vect_len; id++)
	{
		int a = id;
		int b = 2 * id;

		for (int j = 0; j < 10; j++)
		{
			if (id < vect_len / 3)
				continue;
			else
			{
				if (id < vect_len /3 * 2)
					a +=10;
				else
				{
					b += 4;
					if (b == 200)
						break;
				}
				a += 1;
			}
			b += 1;
		}

		if (a != result_v1[id])
		{
			cout << "Test 1 Error at a " << id << " expecting "
				<< a << " getting " << result_v1[id] <<endl;
			flag = false;
		}

		if (b != result_v2[id])
		{
			cout << "Test 1 Error at b " << id << " expecting "
				<< b << " getting " << result_v2[id] <<endl;
			flag = false;
		}
			
	}

	if(flag_1)
		cout << "Verification test passes." <<endl;

	// free device memory
	hipFree( ad );
	hipFree( bd );
	free(vect1);
	free(vect2);
	free(result_v1);
	free(result_v2);
	return EXIT_SUCCESS;
}



